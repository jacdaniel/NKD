#include "hip/hip_runtime.h"

#include <math.h>
#include <malloc.h>
#include <utils.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <laplacian.h>

#define VALUE_INVERSE_LAPLACIAN_1D(x, dx, lambda, mu, value) { \
    if ( x == 0 ) { value = 0.0 ; } \
    else { \
        double wx = 2.0 * PI * (double)x / (double)(2.0*dx); \
        double l1 = -2.0 + 2.0 * cos(wx); \
        double l2 = 6.0 - 8.0 * cos(wx) + 2.0 * cos(2.0 * wx); \
        value = 1.0 / (l1 - mu * l2 - lambda); \
    }\
}

#define VALUE_INVERSE_LAPLACIAN_2D(x, y, dx, dy, lambda, mu, value) { \
    if ( x == 0 && y == 0 ) { value = 0.0; } \
    else {\
        double wx = 2.0 * PI * (double)x / (double)(2.0*dx); \
        double wz = 2.0 * PI * (double)y / (double)(2.0*dy); \
        double l1 = -4.0 + 2.0 * cos(wx) + 2.0 * cos(wz); \
        double l2 = 20.0 - 16.0 * cos(wx) - 16.0 * cos(wz) + 8.0 * cos(wx) * cos(wz) + 2.0 * cos(2.0 * wx) + 2.0 * cos(2.0 * wz); \
        double val = l1 - mu * l2 - lambda; \
        value = 1.0 / val; \
    }\
}


template<typename T>__global__ void laplacian1DFourierGPU_kernel(T* filter, long dx, double lambda, double mu)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < dx)
    {
        double v;
        VALUE_INVERSE_LAPLACIAN_1D(x, dx, lambda, mu, v)
            filter[x] = (T)v;
    }
}

template<typename T>__global__ void laplacian2DFourierGPU_kernel(T* filter, long dx, long dy, double lambda, double mu)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if ( x < dx && y < dy )
    {
        double v;
        VALUE_INVERSE_LAPLACIAN_2D(x, y, dx, dy, lambda, mu, v)
        filter[y * dx + x] = (T)v;
    }
}



template <typename T> int laplacian1DFourierCPUCreate(T** filter, long dx, double lambda, double mu)
{
    long x;

    callocSafe((void**)filter, dx , sizeof(double));
    double v;
    for (x = 0; x < dx; x++)
    {
        VALUE_INVERSE_LAPLACIAN_1D(x, dx, lambda, mu, v)
            (*filter)[x] = (T)v;
    }
  return SUCCESS;
}


template <typename T> int laplacian2DFourierCPUCreate(T **filter, long dx, long dy, double lambda, double mu)
{
  long x, y;
  
  callocSafe((void**)filter, dx*dy, sizeof(double));
  double v;
  for (y = 0; y< dy; y++)
      for (x = 0; x < dx; x++)
      {
          VALUE_INVERSE_LAPLACIAN_2D(x, y, dx, dy, lambda, mu, v)
          (*filter)[y * dx + x] = (T)v;
      }
  return SUCCESS;
}








template<typename T> int laplacianFourierCPUcreate(T** filter, int *size, double lambda, double mu)
{
    callocSafe((void**)filter, size[0]*size[1]*size[2], sizeof(T));
    if (size[1] == 1 && size[2] == 1)
    {
        return laplacian1DFourierCPUCreate<T>(filter, size[0], lambda, mu);
    }
    if (size[2] == 1)
    {
        return laplacian2DFourierCPUCreate<T>(filter, size[0], size[1], lambda, mu);
    }
    else
    {
    }
    return SUCCESS;
}

template<typename T> int laplacianFourierGPUcreate(T** filter, int* size, double lambda, double mu)
{
    hipMalloc((void**)filter, size[0]*size[1]*size[2] * sizeof(T));
    if (size[1] == 1 && size[2] == 1)
    {
        dim3 block(10);
        dim3 grid((size[0] - 1) / block.x + 1);
        laplacian1DFourierGPU_kernel<T> << <grid, block >> > ((T*)(&filter), size[0], lambda, mu);
        return SUCCESS;
    }
    if (size[2] == 1)
    {
        dim3 block(10, 10);
        dim3 grid((size[0] - 1) / block.x + 1, (size[1] - 1) / block.y + 1);
        laplacian2DFourierGPU_kernel<T> << <grid, block >> > ((T*)&filter, size[0], size[1], lambda, mu);
        return SUCCESS;
    }
    else
    {
    }
    return SUCCESS;
}





static int laplacianFourierCPUFormatCreate(void** filter, int* size, double lambda, double mu, int format)
{
    if (format == FORMAT_FLOAT32)
    {
        return laplacianFourierCPUcreate<float>((float**)filter, size, lambda, mu);
    }
    else
    {
        return laplacianFourierCPUcreate<double>((double**)filter, size, lambda, mu);
    }
}

static int laplacianFourierGPUFormatCreate(void** filter, int* size, double lambda, double mu, int format)
{
    if (format == FORMAT_FLOAT32)
    {
        return laplacianFourierGPUcreate<float>((float**)filter, size, lambda, mu);
    }
    else
    {
        return laplacianFourierGPUcreate<double>((double**)filter, size, lambda, mu);
    }
}

int laplacianFourierCreate(void** filter, int* size, double lambda, double mu, int format, int processingMode)
{
    if (filter == nullptr) return FAIL;
    if (processingMode == PROCESSING_MODE_CPU)
    {
        return laplacianFourierCPUFormatCreate(filter, size, lambda, mu, format);
    }
    else
    {
        return laplacianFourierGPUFormatCreate(filter, size, lambda, mu, format);
    }
}


